#include "hip/hip_runtime.h"
//https://aria42.com/blog/2014/12/understanding-lbfgs
#define _CRT_SECURE_NO_WARNINGS
#define USE_LBFGS

#include <omp.h>
#include <assert.h>
#include <iostream>
#include <vector>
#include <deque>
#include <map>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// Using updated (v2) interfaces for CUBLAS and CUSPARSE
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include "hip/hip_runtime_api.h"  // helper function CUDA error checking and initialization
#include "helper_functions.h" // helper for shared functions common to CUDA Samples

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "array2.cuh"

#define THREADS_PER_BLOCK 512

namespace cg = cooperative_groups;

std::vector<int> objEqHeads, gradEqHeads;
std::vector<double> objEqVals, gradEqVals;

enum NodeType {
	NODE_CONST,
	NODE_OPER,
	NODE_VAR
};

enum OpType {
	OP_PLUS = 0,
	OP_MINUS = 1,
	OP_UMINUS = 2,
	OP_TIME = 3,
	OP_DIVIDE = 4,
	OP_SIN,
	OP_COS,
	OP_TG,
	OP_CTG,
	OP_SEC,
	OP_CSC,
	OP_ARCSIN,
	OP_ARCCOS,
	OP_ARCTG,
	OP_ARCCTG,
	OP_POW,
	OP_EXP,
	OP_EEXP,
	OP_SQR,
	OP_SQRT,
	OP_LOG,
	OP_LN,
	OP_NULL = -1
};

typedef struct _EqInfo {
	NodeType _type;
	double _val;
	int _var;
	OpType _op;
	int _left;
	int _right;
} EqInfo;

// enum VarType {
// 	VAR_CONST,
// 	VAR_UNSOLVED,
// 	VAR_SOLVED,
// 	VAR_DELETED,
// 	VAR_FREE
// };

enum OptimType {
	BFGS,
	LBFGS
};

// struct VarInfo {
// 	VarType	_type;
// 	double		_val;

// 	VarInfo(VarType ty, double val) : _type(ty), _val(val) {}
// };

#define epsZero1 1e-20
#define epsZero2 1e-7
#ifndef M_PI_2
#define M_PI_2 (1.57079632679489661923)
#endif


#define		BFGS_MAXIT	500
#define		BFGS_STEP	0.1

static int _GetMaxIt()
{
	return BFGS_MAXIT;
}

static double _GetStep()
{
	return BFGS_STEP;
}

__host__ __device__ static double _GetEps()
{
	return 0.01;
}

static void _ConstructVarTab(std::vector<double>& vars, std::vector<int>& varMap, std::vector<int>& revMap);
static void _ConstructObjEqTab(std::vector<EqInfo>& eqs, int& numEqs, const std::vector<int>& revMap);
static void _ConstructGradEqTab(std::vector<EqInfo>& eqs, int& numEqs, const std::vector<int>& revMap);
static void _ScatterVarTab(std::vector<double>& x, std::vector<int>& varMap);

static void _VecCopy(std::vector<double>& dst, const std::vector<double>& src)
{
	int n = src.size();
	for (int i = 0; i < n; i++)
		dst[i] = src[i];
}

static void _VecSub(const std::vector<double>& a, const std::vector<double>& b, std::vector<double>& ret)
{
	int n = a.size();
	for (int i = 0; i < n; i++)
		ret[i] = a[i] - b[i];
}

static double _VecDot(const std::vector<double>& a, const std::vector<double>& b)
{
	double s = 0;
	int n = a.size();
	for (int i = 0; i < n; i++)
		s += a[i] * b[i];

	return s;
}

static void _VecMult(std::vector<double>& v, double t)
{
	int n = v.size();
	for (int i = 0; i < n; i++)
		v[i] *= t;
}

static double _VecAdd(const std::vector<double>& a, const std::vector<double>& b, std::vector<double>& ret)
{
	int n = a.size();
	for (int i = 0; i < n; i++)
		ret[i] = a[i] + b[i];
}

static void _VecAxPy(const std::vector<double>& a, double t, const std::vector<double>& b, std::vector<double>& ret)
{
	int n = a.size();
	for (int i = 0; i < n; i++)
		ret[i] = a[i] + b[i] * t;
}

static double _VecLen(const std::vector<double>& v)
{
	return sqrt(_VecDot(v, v));
}

static void _VecNorm(std::vector<double>& v)
{
	double tmp = _VecLen(v);
	if (tmp > 0.0) {
		_VecMult(v, 1.0 / tmp);
	}
}

static void 
_CalcEqNew2(const std::vector<double>& x, const std::vector<EqInfo>& etab, int st, int ed, std::vector<double>& vtab)
{
	for (int i = ed - 1; i >= st; i--) {
		const EqInfo& eq = etab[i];
		switch (eq._type) {
		case NODE_CONST:
			vtab[i] = eq._val;
			break;

		case NODE_VAR: {
			int idx = eq._var;
			vtab[i] = x[idx];
			break;
		}

		case NODE_OPER: {
			double left = vtab[eq._left];
			double right = vtab[eq._right];
			switch (eq._op) {
			case	OP_PLUS:
				vtab[i] = (left + right);
				break;
			case	OP_MINUS:
				vtab[i] = (left - right);
				break;
			case	OP_UMINUS:
				vtab[i] = -right;
				break;
			case	OP_TIME:
				vtab[i] = (left * right);
				break;
			case	OP_DIVIDE:
				vtab[i] = (left / right);
				break;
			case	OP_SIN:
				vtab[i] = (sin(left));
				break;
			case	OP_COS:
				vtab[i] = (cos(left));
				break;
			case	OP_TG:
				vtab[i] = (tan(left));
				break;
			case	OP_CTG:
				vtab[i] = (1.0 / tan(left));
				break;
			case	OP_SEC:
				vtab[i] = (1.0 / cos(left));
				break;
			case	OP_CSC:
				vtab[i] = (1.0 / sin(left));
				break;
			case	OP_ARCSIN:
				vtab[i] = (asin(left));
				break;
			case	OP_ARCCOS:
				vtab[i] = (acos(left));
				break;
			case	OP_ARCTG:
				vtab[i] = (atan(left));
				break;
			case	OP_ARCCTG:
				vtab[i] = (atan(-left) + M_PI_2);
				break;
			case	OP_POW:
				vtab[i] = (pow(left, right));
				break;
			case	OP_EEXP:
				vtab[i] = (exp(left));
				break;
			case	OP_EXP:
				vtab[i] = (exp(left * log(right)));
				break;
			case	OP_LN:
				vtab[i] = (log(left));
				break;
			case	OP_LOG:
				vtab[i] = (log(right) / log(left));
				break;
			case	OP_SQR:
				vtab[i] = (left * left);
				break;
			case	OP_SQRT:
				vtab[i] = (sqrt(left));
				break;
			default:
				fprintf(stderr, "Unknown operator in EsCalcTree()\n");
				assert(0);
			}
		}
		}

	}
}

static double _CalcEqNew1(const std::vector<double>& x, const EqInfo& eq, const std::vector<EqInfo>& etab,
	int item, const std::vector<int> &htab, int allNum, std::vector<double> &vtab)
{
	int ed = item < 0 ? allNum : htab[item + 1];
	int st = item < 0 ? htab[-item] : htab[item];
	_CalcEqNew2(x, etab,  st, ed, vtab);

	switch (eq._type) {
	case NODE_OPER: {
		double left = vtab[eq._left];
		double right = vtab[eq._right];
		switch (eq._op) {
		case	OP_PLUS:
			return(left + right);
		case	OP_MINUS:
			return(left - right);
		case	OP_UMINUS:
			return(-right);
		case	OP_TIME:
			return(left * right);
		case	OP_DIVIDE:
			return(left / right);
		case	OP_SIN:
			return(sin(left));
		case	OP_COS:
			return(cos(left));
		case	OP_TG:
			return(tan(left));
		case	OP_CTG:
			return(1.0 / tan(left));
		case	OP_SEC:
			return(1.0 / cos(left));
		case	OP_CSC:
			return(1.0 / sin(left));
		case	OP_ARCSIN:
			return(asin(left));
		case	OP_ARCCOS:
			return(acos(left));
		case	OP_ARCTG:
			return(atan(left));
		case	OP_ARCCTG:
			return(atan(-left) + M_PI_2);
		case	OP_POW:
			return(pow(left, right));
		case	OP_EEXP:
			return(exp(left));
		case	OP_EXP:
			return(exp(left * log(right)));
		case	OP_LN:
			return(log(left));
		case	OP_LOG:
			return(log(right) / log(left));
		case	OP_SQR:
			return(left * left);
		case	OP_SQRT:
			return(sqrt(left));
		default:
			fprintf(stderr, "Unknown operator in EsCalcTree()\n");
			assert(0);
			return  (0.0);
		}
	}
	}

	assert(0);
	return 0;
}

static double _CalcEq(const std::vector<double>& x, const EqInfo& eq, const std::vector<EqInfo>& etab)
{
	double left, right;

	switch (eq._type) {
	case NODE_CONST:
		return(eq._val);
		break;

	case NODE_VAR: {
		int idx = eq._var;
		return x[idx];
		break;
	}

	case NODE_OPER: {
		left = _CalcEq(x, etab[eq._left], etab);
		right = _CalcEq(x, etab[eq._right], etab);
		switch (eq._op) {
		case	OP_PLUS:
			return(left + right);
		case	OP_MINUS:
			return(left - right);
		case	OP_UMINUS:
			return(-right);
		case	OP_TIME:
			return(left * right);
		case	OP_DIVIDE:
			return(left / right);
		case	OP_SIN:
			return(sin(left));
		case	OP_COS:
			return(cos(left));
		case	OP_TG:
			return(tan(left));
		case	OP_CTG:
			return(1.0 / tan(left));
		case	OP_SEC:
			return(1.0 / cos(left));
		case	OP_CSC:
			return(1.0 / sin(left));
		case	OP_ARCSIN:
			return(asin(left));
		case	OP_ARCCOS:
			return(acos(left));
		case	OP_ARCTG:
			return(atan(left));
		case	OP_ARCCTG:
			return(atan(-left) + M_PI_2);
		case	OP_POW:
			return(pow(left, right));
		case	OP_EEXP:
			return(exp(left));
		case	OP_EXP:
			return(exp(left * log(right)));
		case	OP_LN:
			return(log(left));
		case	OP_LOG:
			return(log(right) / log(left));
		case	OP_SQR:
			return(left * left);
		case	OP_SQRT:
			return(sqrt(left));
		default:
			fprintf(stderr, "Unknown operator in EsCalcTree()\n");
			assert(0);
			return  (0.0);
		}
	}
	}

	assert(0);
	return 0;
}

static double _CalcObj(const std::vector<double>& x,
	const std::vector<EqInfo>& eqs, int eqNum)
{
	std::vector<double> tmp;
	tmp.resize(eqNum);

	for (int i = 0; i < eqNum; i++) {
		//double v1 = _CalcEq(x, eqs[i], eqs);
		double v2 = _CalcEqNew1(x, eqs[i], eqs, i == eqNum - 1 ? -i : i, objEqHeads, objEqVals.size(), objEqVals);
		//assert(v1 == v2);
		tmp[i] = v2;
	}

	return _VecDot(tmp, tmp);
}

static void _CalcGrad(const std::vector<double>& x, std::vector<double>& g,
	const std::vector<EqInfo>& eqs)
{
	int n = x.size();
	for (int i = 0; i < n; i++) {
		//double v1 = _CalcEq(x, eqs[i], eqs);
		double v2 = _CalcEqNew1(x, eqs[i], eqs, i == n - 1 ? -i : i, gradEqHeads, gradEqVals.size(), gradEqVals);
		//assert(v1 == v2);
		g[i] = v2;
	}
}

static double _CalcObj(const std::vector<double>& x0, double h, const std::vector<double>& p,
	const std::vector<EqInfo>& eqs, int eqNum)
{
	std::vector<double> xt;
	xt.resize(x0.size());
	_VecAxPy(x0, h, p, xt);
	return _CalcObj(xt, eqs, eqNum);
}

static void _CalcyTH(const std::vector<double>& y, const array2<double>& H, std::vector<double>& yTH)
{
	int	i, j;
	int n = y.size();

	std::fill(yTH.begin(), yTH.end(), 0.0);
	for (j = 0; j < n; j++)
		for (i = 0; i < n; i++) {
			yTH[i] += (y[j] * H(j, i));
		}
}

static void _CalcHy(const array2<double>& H, const std::vector<double>& y, std::vector<double>& Hy)
{
	int	i, j;
	int n = y.size();

	for (i = 0; i < n; i++) {
		Hy[i] = 0.0;
		for (j = 0; j < n; j++)
			Hy[i] += (y[j] * H(i, j));
	}
}

static void _Calcp(const array2<double>& H, const std::vector<double>& g, std::vector<double>& p)
{
	_CalcHy(H, g, p);

	int n = p.size();
	while (n--)
		p[n] = -p[n];
}

#define BFGS_MAXBOUND	1e+10
static void _DetermineInterval(
	const std::vector<double>& x0, double h, const std::vector<double>& p,
	double* left, double* right,
	const std::vector<EqInfo>& eqs, int eqNum)
{
	double	A, B, C, D, u, v, w, s, r;

	A = _CalcObj(x0, 0.0, p, eqs, eqNum);
	B = _CalcObj(x0, h, p, eqs, eqNum);
	if (B > A) {
		s = -h;
		C = _CalcObj(x0, s, p, eqs, eqNum);
		if (C > A) {
			*left = -h;
			*right = h;
			return;
		}
		B = C;
	}
	else {
		s = h;
	}
	u = 0.0;
	v = s;
	while (1) {
		s += s;
		if (fabs(s) > BFGS_MAXBOUND) {
			*left = *right = 0.0;
			return;
		}
		w = v + s;
		C = _CalcObj(x0, w, p, eqs, eqNum);
		if (C >= B)
			break;
		u = v;
		A = B;
		v = w;
		B = C;
	}
	r = (v + w) * 0.5;
	D = _CalcObj(x0, r, p, eqs, eqNum);
	if (s < 0.0) {
		if (D < B) {
			*left = w;
			*right = v;
		}
		else {
			*left = r;
			*right = u;
		}
	}
	else {
		if (D < B) {
			*left = v;
			*right = w;
		}
		else {
			*left = u;
			*right = r;
		}
	}
}

static void _GodenSep(
	const std::vector<double>& x0, const std::vector<double>& p,
	double left, double right, std::vector<double>& x,
	const std::vector<EqInfo>& eqs, int eqNum)
{
	static double	beta = 0.61803398874989484820;
	double			t1, t2, f1, f2;

	t2 = left + beta * (right - left);
	f2 = _CalcObj(x0, t2, p, eqs, eqNum);
ENTRY1:
	t1 = left + right - t2;
	f1 = _CalcObj(x0, t1, p, eqs, eqNum);
ENTRY2:
	if (fabs(t1 - t2) < epsZero2) {
		t1 = (t1 + t2) / 2.0;
		//printf("LineSearch t = %lf\n", t1*10000);

		_VecAxPy(x0, t1, p, x);
		return;
	}
	if ((fabs(left) > BFGS_MAXBOUND) || (fabs(left) > BFGS_MAXBOUND))
		return;
	if (f1 <= f2) {
		right = t2;
		t2 = t1;
		f2 = f1;
		goto ENTRY1;
	}
	else {
		left = t1;
		t1 = t2;
		f1 = f2;
		t2 = left + beta * (right - left);
		f2 = _CalcObj(x0, t2, p, eqs, eqNum);
		goto ENTRY2;
	}
}

static void _LinearSearch(
	const std::vector<double>& x0,
	const std::vector<double>& p,
	double h,
	std::vector<double>& x,
	const std::vector<EqInfo>& eqs,
	int eqNum)
{
	double	left, right;

	_DetermineInterval(x0, h, p, &left, &right, eqs, eqNum);
	if (left == right)
		return;

	//printf("%lf, %lf\n", left, right);
	_GodenSep(x0, p, left, right, x, eqs, eqNum);
}

#define	H_EPS1	1e-5
#define	H_EPS2	1e-5
#define	H_EPS3	1e-4

static bool _HTerminate(
	const std::vector<double>& xPrev,
	const std::vector<double>& xNow,
	double fPrev, double fNow,
	const std::vector<double>& gNow)
{
	double	ro;
	std::vector<double> xDif(xNow.size());

	if (_VecLen(gNow) >= H_EPS3)
		return false;

	_VecSub(xNow, xPrev, xDif);
	ro = _VecLen(xPrev);
	if (ro < H_EPS2)
		ro = 1.0;
	ro *= H_EPS1;
	if (_VecLen(xDif) >= ro)
		return false;

	ro = fabs(fPrev);
	if (ro < H_EPS2)
		ro = 1.0;
	ro *= H_EPS1;
	fNow -= fPrev;
	if (fabs(fNow) >= ro)
		return false;

	return true;
}

void
AnalysisEqs(const std::vector<EqInfo>& eqTab, int eqNum, std::vector<int>& eqHeads)
{
	eqHeads.resize(eqNum);
	for (int i = 0; i < eqNum; i++) {
		const EqInfo& eq = eqTab[i];
		int left = eq._left;
		int right = eq._right;

		eqHeads[i] = left;
	}
}

__device__ inline double evaluate_operator(int op, double left, double right) {
    switch (op) {
        case OP_PLUS:   return left + right;
        case OP_MINUS:  return left - right;
        case OP_UMINUS: return -right;
        case OP_TIME:   return left * right;
        case OP_DIVIDE: return left / right;
        case OP_SIN:    return sin(left);
        case OP_COS:    return cos(left);
        case OP_TG:     return tan(left);
        case OP_CTG:    return 1.0 / tan(left);
        case OP_SEC:    return 1.0 / cos(left);
        case OP_CSC:    return 1.0 / sin(left);
        case OP_ARCSIN: return asin(left);
        case OP_ARCCOS: return acos(left);
        case OP_ARCTG:  return atan(left);
        case OP_ARCCTG: return atan(-left) + M_PI_2;
        case OP_POW:    return pow(left, right);
        case OP_EEXP:   return exp(left);
        case OP_EXP:    return exp(left * log(right));
        case OP_LN:     return log(left);
        case OP_LOG:    return log(right) / log(left);
        case OP_SQR:    return left * left;
        case OP_SQRT:   return sqrt(left);
        default:
            printf("Unknown operator in evaluate_operator\n");
            assert(0);
            return 0.0;
    }
}

__device__ void gpuSpVM(double *matrix, double *vector, double *result, 
                                        int rows, int cols, double alpha,
										const cg::grid_group &grid) {
    for (int i = grid.thread_rank(); i < cols; i += grid.size()) {
        double output = 0.0;
        for (int j = 0; j < rows; j++) 
		{
        	output += alpha * vector[j] * matrix[cols * j + i];
        }

        result[i] = output;
		// printf("yTH[%d] = %f\n", i, result[i]);
    }
}

__device__ void gpuSpMV(double *matrix, double *vector, double *result, 
                                        int rows, int cols, double alpha,
										const cg::grid_group &grid) {
    for (int i = grid.thread_rank(); i < rows; i += grid.size()) {
        double output = 0.0;
        for (int j = 0; j < cols; j++) 
		{
        	output += alpha * vector[j] * matrix[cols * i + j];
        }

        result[i] = output;
    }
}

__device__ void gpuScaleVector(double *y, double scale, int size,
                            const cg::grid_group &grid) {
    for (int i = grid.thread_rank(); i < size; i += grid.size()) {
        y[i] = scale * y[i];
    }
}

__device__ void gpuSaxpy(double *x, double *y, double *r, double a, int size,
                         const cg::grid_group &grid) {
    for (int i = grid.thread_rank(); i < size; i += grid.size()) {
        r[i] = a * x[i] + y[i];
		// printf("r[%d] = %f\n", i, r[i]);
    }
}

__device__ void gpuDotProduct(double *vecA, double *vecB, double *result,
                              int size, const cg::thread_block &cta,
                              const cg::grid_group &grid) {
    extern __shared__ double tmp[];

    double temp_sum = 0.0;
    for (int i = grid.thread_rank(); i < size; i += grid.size()) {
        temp_sum += vecA[i] * vecB[i];
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
        tmp[tile32.meta_group_rank()] = temp_sum;
    }

    cg::sync(cta);

    if (tile32.meta_group_rank() == 0) {
        temp_sum = tile32.thread_rank() < tile32.meta_group_size() ? tmp[tile32.thread_rank()] : 0.0;
        temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

        if (tile32.thread_rank() == 0) {
        	atomicAdd(result, temp_sum);
        }
    }
}

__device__ void gpuCopyVector(double *srcA, double *destB, int size,
                                const cg::grid_group &grid) {
    for (int i = grid.thread_rank(); i < size; i += grid.size()) {
        destB[i] = srcA[i];
    }
}

__device__ void gpuScaleVectorAndSaxpy(const double *x, double *y, double a, double scale, int size,
                            const cg::grid_group &grid) {
    for (int i = grid.thread_rank(); i < size; i += grid.size()) {
        y[i] = a * x[i] + scale * y[i];
    }
}

__device__ void gpuHUpdate(double *H, double *Hy, double *yTH, double tmp, double sy, double *s, int n, const cg::grid_group &grid) 
{
	for (int i = grid.thread_rank(); i < n * n; i += grid.size()) {
        int row = i / n;
		int col = i % n;
		H[i] += (((tmp * s[row] * s[col]) - Hy[row] * s[col] -
					s[row] * yTH[col]) / sy);
	}
}

__device__ bool gpuHTerminate(
	double* xPrev, double* xNow,
	double fPrev, double fNow,
	double* gNow, double* xDif, 
	double *dot_result, int n,
	const cg::thread_block &cta,
    const cg::grid_group &grid)
{
	double	ro;
	double alpham1 = -1.0;

	if(threadIdx.x == 0 && blockIdx.x == 0) *dot_result = 0.0;
	cg::sync(grid);

	gpuDotProduct(gNow, gNow, dot_result, n, cta, grid);
	cg::sync(grid);

	if(sqrt(*dot_result) >= H_EPS3)
		return false;

	gpuSaxpy(xPrev, xNow, xDif, alpham1, n, grid);
	cg::sync(grid);


	if(threadIdx.x == 0 && blockIdx.x == 0) *dot_result = 0.0;
	cg::sync(grid);

	gpuDotProduct(xPrev, xPrev, dot_result, n, cta, grid);
	cg::sync(grid);

	ro = sqrt(*dot_result);

	if (ro < H_EPS2)
		ro = 1.0;
	ro *= H_EPS1;

	if(threadIdx.x == 0 && blockIdx.x == 0) *dot_result = 0.0;
	cg::sync(grid);

	gpuDotProduct(xDif, xDif, dot_result, n, cta, grid);
	cg::sync(grid);

	if (sqrt(*dot_result) >= ro)
		return false;

	ro = fabs(fPrev);
	if (ro < H_EPS2)
		ro = 1.0;
	ro *= H_EPS1;

	double tmp = abs(fNow - fPrev);
	if (fabs(tmp) >= ro)
		return false;

	return true;
}

__device__ void gpuCalcEq(double* x, EqInfo* etab, int st, int ed, double* vtab)
{
	for (int i = ed - 1; i >= st; i--) 
	{
		const EqInfo eq = etab[i];
		if (eq._type == NODE_CONST)
		{
			vtab[i] = eq._val;
		} 
		else if (eq._type == NODE_VAR)
		{
			int idx = eq._var;
			vtab[i] = x[idx];
		}
		else if (eq._type == NODE_OPER)
		{
			double left = vtab[eq._left];
			double right = vtab[eq._right];
			vtab[i] = evaluate_operator(eq._op, left, right);
		}
	}
}


__global__ void gpuCalcGrad(double* x, double* g, int n, int allnum, int* gradEqHeads,
	EqInfo* eqs, double* gradEqVal)
{
	int gthIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int gridSize = gridDim.x * blockDim.x;

	for (int i = gthIdx; i < n; i += gridSize)
	{
		int item = i == n - 1 ? -i : i;
		int ed = item < 0 ? allnum : gradEqHeads[item + 1];
		int st = item < 0 ? gradEqHeads[-item] : gradEqHeads[item];

		gpuCalcEq(x, eqs, st, ed, gradEqVal);

		EqInfo eq = eqs[i];
		if(eq._type == NODE_OPER)
		{
			double left = gradEqVal[eq._left];
			double right = gradEqVal[eq._right];
			g[i] = evaluate_operator(eq._op, left, right);
		}
		else
		{
			g[i] = 0.0;
			assert(0);
		}
	}
}

__global__ void gpuCalcObj(double* x, double* tmp, int eqNum, int allNum, int* objEqHeads,
	EqInfo* eqs, double* objEqVal, double* result)
{
	cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
	
	if(threadIdx.x == 0 && blockIdx.x == 0) *result = 0.0;
	cg::sync(grid);
	
	for (int i = grid.thread_rank(); i < eqNum; i += grid.size())
	{
		int item = i == eqNum - 1 ? -i : i;
		int ed = item < 0 ? allNum : objEqHeads[item + 1];
		int st = item < 0 ? objEqHeads[-item] : objEqHeads[item];

		gpuCalcEq(x, eqs, st, ed, objEqVal);

		EqInfo eq = eqs[i];
		if(eq._type == NODE_OPER)
		{
			double left = objEqVal[eq._left];
			double right = objEqVal[eq._right];
			tmp[i] = evaluate_operator(eq._op, left, right);
		}
		else
		{
			tmp[i] = 0.0;
			assert(0);
		}
	}

	cg::sync(grid);

	gpuDotProduct(tmp, tmp, result, eqNum, cta, grid);
	cg::sync(grid);
}

__device__ void gpuCalcGradx(double* x, double* g, int n, int allnum, int* gradEqHeads,
	EqInfo* eqs, double* gradEqVal, const cg::grid_group &grid)
{
	int gthIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int gridSize = gridDim.x * blockDim.x;

	for (int i = grid.thread_rank(); i < n; i += grid.size())
	{
		int item = i == n - 1 ? -i : i;
		int ed = item < 0 ? allnum : gradEqHeads[item + 1];
		int st = item < 0 ? gradEqHeads[-item] : gradEqHeads[item];

		gpuCalcEq(x, eqs, st, ed, gradEqVal);

		EqInfo eq = eqs[i];
		if(eq._type == NODE_OPER)
		{
			double left = gradEqVal[eq._left];
			double right = gradEqVal[eq._right];
			g[i] = evaluate_operator(eq._op, left, right);
		}
		else
		{
			g[i] = 0.0;
			assert(0);
		}
	}
}

__device__ void gpuCalcObjOffset(double *xt, double* result, double* tmp, int allNum, EqInfo* eqs, int* objEqHeads, double* objEqVal, int eqNum, int n, cooperative_groups::thread_block& cta, cooperative_groups::grid_group& grid)
{

	if(threadIdx.x == 0 && blockIdx.x == 0) *result = 0.0;
	cg::sync(grid);
	
	for (int i = grid.thread_rank(); i < eqNum; i += grid.size())
	{
		int item = i == eqNum - 1 ? -i : i;
		int ed = item < 0 ? allNum : objEqHeads[item + 1];
		int st = item < 0 ? objEqHeads[-item] : objEqHeads[item];

		gpuCalcEq(xt, eqs, st, ed, objEqVal);

		EqInfo eq = eqs[i];
		if(eq._type == NODE_OPER)
		{
			double left = objEqVal[eq._left];
			double right = objEqVal[eq._right];
			tmp[i] = evaluate_operator(eq._op, left, right);
		}
		else
		{
			tmp[i] = 0.0;
			assert(0);
		}
	}

	cg::sync(grid);

	gpuDotProduct(tmp, tmp, result, eqNum, cta, grid);
	cg::sync(grid);
}

__global__ void gpuDetermineInterval(
    double* x0, double h, double* p, double* xt, int n,
    double* left, double* right, double* result, double* tmp,
    EqInfo* eqs, int eqNum, int allNum, int* objEqHeads,
	double* objEqVal)
{
    double A, B, C, D, u, v, w, s, r;
	
	cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    // Initialize A and B with values from _CalcObj
	// xt = x0 + h * p
	gpuSaxpy(p, x0, xt, h, n, grid);
	cg::sync(grid);

	gpuCalcObjOffset(x0, result, tmp, allNum, eqs, objEqHeads, objEqVal, eqNum, n, cta, grid);

	A = *result;
	cg::sync(grid);
	
	gpuCalcObjOffset(xt, result, tmp, allNum, eqs, objEqHeads, objEqVal, eqNum, n, cta, grid);
	
	B = *result;
	cg::sync(grid);
	
    // A = _CalcObj(x0, 0.0, p, eqs, eqNum);
    // B = _CalcObj(x0, h, p, eqs, eqNum);
    if (B > A) {
        s = -h;
		gpuSaxpy(p, x0, xt, s, n, grid);
		cg::sync(grid);
        // C = _CalcObj(x0, s, p, eqs, eqNum);
		gpuCalcObjOffset(xt, result, tmp, allNum, eqs, objEqHeads, objEqVal, eqNum, n, cta, grid);

		C = *result;
		cg::sync(grid);
		
        if (C > A) {
            *left = -h;
            *right = h;
            return;
        }
        B = C;
    }
    else {
        s = h;
    }

    // Initialize u and v
    u = 0.0;
    v = s;

    while (1) {
        s += s;
        if (fabs(s) > BFGS_MAXBOUND) {
            *left = *right = 0.0;
            return;
        }

        // Calculate w and evaluate C at w
        w = v + s;
		gpuSaxpy(p, x0, xt, w, n, grid);
		cg::sync(grid);
		gpuCalcObjOffset(xt, result, tmp, allNum, eqs, objEqHeads, objEqVal, eqNum, n, cta, grid);

		C = *result;
		cg::sync(grid);
        // C = _CalcObj(x0, w, p, eqs, eqNum);

        if (C >= B)
            break;

        // Update values for u, A, v, and B
        u = v;
        A = B;
        v = w;
        B = C;
    }

    // Midpoint calculation for r
    r = (v + w) * 0.5;
	gpuSaxpy(p, x0, xt, r, n, grid);
	cg::sync(grid);
	gpuCalcObjOffset(xt, result, tmp, allNum, eqs, objEqHeads, objEqVal, eqNum, n, cta, grid);
	D = *result;
	cg::sync(grid);
    // D = _CalcObj(x0, r, p, eqs, eqNum);

    // Set interval bounds based on s
    if (s < 0.0) {
        if (D < B) {
            *left = w;
            *right = v;
        }
        else {
            *left = r;
            *right = u;
        }
    }
    else {
        if (D < B) {
            *left = v;
            *right = w;
        }
        else {
            *left = u;
            *right = r;
        }
    }
}

__global__ void gpuGodenSep(
	double* x0, double* p, double* xt, double* x, int n,
    double *left, double *right, double* result, double* tmp,
    EqInfo* eqs, int eqNum, int allNum, int* objEqHeads,
	double* objEqVal)
{
	static double	beta = 0.61803398874989484820;
	double			t1, t2, f1, f2;
	bool mark = true;

	cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

	if(*left == *right)
		return;

	t2 = *left + beta * (*right - *left);
	gpuSaxpy(p, x0, xt, t2, n, grid);
	cg::sync(grid);
	gpuCalcObjOffset(xt, result, tmp, allNum, eqs, objEqHeads, objEqVal, eqNum, n, cta, grid);
	f2 = *result;
	cg::sync(grid);
	while(1)
	{
		if(mark)
		{
			t1 = *left + *right - t2;
			gpuSaxpy(p, x0, xt, t1, n, grid);
			cg::sync(grid);
			gpuCalcObjOffset(xt, result, tmp, allNum, eqs, objEqHeads, objEqVal, eqNum, n, cta, grid);
			f1 = *result;
			cg::sync(grid);
		}

		if (fabs(t1 - t2) < epsZero2) 
		{
			t1 = (t1 + t2) / 2.0;
			gpuSaxpy(p, x0, x, t1, n, grid);
			cg::sync(grid);
			break;
		}
		if ((fabs(*left) > BFGS_MAXBOUND) || (fabs(*left) > BFGS_MAXBOUND))
			break;
		if (f1 <= f2) 
		{
			*right = t2;
			t2 = t1;
			f2 = f1;
			mark = true;
		}
		else 
		{
			*left = t1;
			t1 = t2;
			f1 = f2;
			t2 = *left + beta * (*right - *left);
			gpuSaxpy(p, x0, xt, t2, n, grid);
			cg::sync(grid);
			gpuCalcObjOffset(xt, result, tmp, allNum, eqs, objEqHeads, objEqVal, eqNum, n, cta, grid);
			f2 = *result;
			cg::sync(grid);
			mark = false;
		}
	}
}

__global__ void gpuInitHp(double* H, double* p, double* gPrev, double* result, int n)
{
	cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
	
	double alpham1 = -1.0;
	
	for(int i = grid.thread_rank(); i < n; i += grid.size())
	{
		H[i * n + i] = 1.0;
		p[i] = alpham1 * gPrev[i];
	}
	
	cg::sync(grid);
	
	gpuDotProduct(p, p, result, n, cta, grid);
	cg::sync(grid);
	gpuScaleVector(p, 1.0/sqrt(*result), n, grid);
	cg::sync(grid);
}

extern "C" __global__ void BFGSMultiply(
    double* gPrev, double* gNow, double* xPrev, double* xNow, double* H,
    double* p, double* yTH, double* Hy, double* s, double* y, double fPrev, double fNow,
    double* sy, double* dot_result, int n, int allnum, int *gradEqHeads, EqInfo *gradEqls, double *gradEqVals) 
{
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    double alpha = 1.0;
    double alpham1 = -1.0;
	
	// gpuCalcGradx(xNow, gNow, n, allnum, gradEqHeads, gradEqls, gradEqVals, grid);
	
	// cg::sync(grid);

	bool con = gpuHTerminate(xPrev, xNow, fPrev, fNow, gNow, s, dot_result, n, cta, grid);
	
	if(con)
	{
		*sy = 0;
		return;
	}

	// Compute y = gNow - gPrev
	gpuSaxpy(gPrev, gNow, y, alpham1, n, grid);
	cg::sync(grid);

	// Compute s = xNow - xPrev
	gpuSaxpy(xPrev, xNow, s, alpham1, n, grid);
	cg::sync(grid);

	// Compute sy = dot(s, y)
	gpuDotProduct(s, y, sy, n, cta, grid);
	cg::sync(grid);

	// Proceed only if sy is above a certain threshold (epsZero1)
	if (fabs(*sy) >= epsZero1) {
		// Compute yTH = H * y
		gpuSpVM(H, y, yTH, n, n, alpha, grid);
		cg::sync(grid);

		// Compute Hy = H * y
		gpuSpMV(H, y, Hy, n, n, alpha, grid);
		cg::sync(grid);

		// Initialize dot_result to zero
		if (threadIdx.x == 0 && blockIdx.x == 0) *dot_result = 0;
		cg::sync(grid);

		// Compute dot_result = dot(yTH, y)
		gpuDotProduct(yTH, y, dot_result, n, cta, grid);
		cg::sync(grid);
		// Update H matrix with computed values
		double tmp = 1.0 + *dot_result / *sy;

		gpuHUpdate(H, Hy, yTH, tmp, *sy, s, n, grid);
		cg::sync(grid);

		// Compute p = -H * gNow
		gpuSpMV(H, gNow, p, n, n, alpham1, grid);
		cg::sync(grid);

		// Re-initialize dot_result to zero before re-use
		if (threadIdx.x == 0 && blockIdx.x == 0) *dot_result = 0;
		cg::sync(grid);

		// Compute the norm of p
		gpuDotProduct(p, p, dot_result, n, cta, grid);
		cg::sync(grid);

		// Normalize p to unit length
		gpuScaleVector(p, 1.0 / sqrt(*dot_result), n, grid);
		cg::sync(grid);
	}
}

int BFGSSolveEqs()
{
	double eps = _GetEps()*_GetEps();
	int itMax = _GetMaxIt();

	double step = _GetStep();

	std::vector<double> xNow, xKeep;
	std::vector<int> varMap, revMap;
	std::vector<EqInfo> objEqs;
	int numObjEqs;
	std::vector<EqInfo> gradEqs;
	int numGradEqs;

	hipDeviceProp_t deviceProp;
	int devID = 0;
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

	if (!deviceProp.managedMemory) {
		// This sample requires being run on a device that supports Unified Memory
		fprintf(stderr, "Unified Memory not supported on this device\n");
		exit(EXIT_WAIVED);
	}

	// This sample requires being run on a device that supports Cooperative Kernel
	// Launch
	if (!deviceProp.cooperativeLaunch) {
		printf(
				"\nSelected GPU (%d) does not support Cooperative Kernel Launch, "
				"Waiving the run\n",
				devID);
		exit(EXIT_WAIVED);
	}

	FILE* fp = fopen("D:\\yzj\\bfgs\\data\\bfgs-large.dat", "rb");
	if (fp == NULL) {
		printf("bfgs.dat failed to open for read.\n");
		return false;
	}

	{
		double t0 = omp_get_wtime();

		int nx;
		fread(&nx, sizeof(int), 1, fp);
		xNow.resize(nx);
		fread(xNow.data(), sizeof(double), nx, fp);

		int n1, no;
		fread(&n1, sizeof(int), 1, fp);
		fread(&no, sizeof(int), 1, fp);
		numObjEqs = no;
		objEqs.resize(n1);
		fread(objEqs.data(), sizeof(EqInfo), n1, fp);

		int ng;
		fread(&ng, sizeof(int), 1, fp);
		gradEqs.resize(ng);
		fread(gradEqs.data(), sizeof(EqInfo), ng, fp);
		numGradEqs = ng;

		int nk;
		fread(&nk, sizeof(int), 1, fp);
		assert(nk == nx);
		xKeep.resize(nk);
		fread(xKeep.data(), sizeof(double), nk, fp);

		double dt = omp_get_wtime() - t0;
		printf("###Data loading used %2.5f s ...\n", dt);

		//to remove recursive eval
		AnalysisEqs(objEqs, numObjEqs, objEqHeads);
		objEqVals.resize(objEqs.size());
		AnalysisEqs(gradEqs, nx, gradEqHeads);
		gradEqVals.resize(gradEqs.size());
	}

	double t0 = omp_get_wtime();
	//Do optimization
	double fNow = 0, fPrev = 0;;
	int n = xNow.size();
	int k = 0;//useless?
	int itCounter = 0;

	std::vector<double> gPrev, gNow, xPrev, p, y, s, yTH, Hy;

	array2<double> H;

	xPrev = xNow;
	gPrev.resize(n);
	gNow.resize(n);
	p.resize(n);
	y.resize(n);
	s.resize(n);
	yTH.resize(n);
	Hy.resize(n);
	H.resize(n, n);

	double *d_gnow, *d_gprev, *d_xnow, *d_xprev, *left, *right, *d_xt, *d_s, *d_y, *d_yTH, *d_Hy, *d_p, *d_H, *d_sy, *d_dot_result, *d_gradEqVals, *d_objEqVals, *d_fprev, *d_fnow, *tmp;
	EqInfo* d_gradEqls, *d_objEqls;
	int* d_gradEqHeads, *d_objEqHeads;
	checkCudaErrors(hipMalloc(&d_gnow, sizeof(double) * n));
	checkCudaErrors(hipMalloc(&d_gprev, sizeof(double) * n));
	checkCudaErrors(hipMalloc(&d_xnow, sizeof(double) * n));
	checkCudaErrors(hipMalloc(&d_xprev, sizeof(double) * n));
	checkCudaErrors(hipMalloc(&d_xt, sizeof(double) * n));
	checkCudaErrors(hipMalloc(&left, sizeof(double)));
	checkCudaErrors(hipMalloc(&right, sizeof(double)));
	checkCudaErrors(hipMalloc(&d_s, sizeof(double) * n));
	checkCudaErrors(hipMalloc(&d_y, sizeof(double) * n));
	checkCudaErrors(hipMalloc(&d_yTH, sizeof(double) * n));
	checkCudaErrors(hipMalloc(&d_Hy, sizeof(double) * n));
	checkCudaErrors(hipMalloc(&d_p, sizeof(double) * n));
	checkCudaErrors(hipMalloc(&d_H, sizeof(double) * n * n));
	checkCudaErrors(hipMalloc(&d_sy, sizeof(double)));
	checkCudaErrors(hipMalloc(&d_dot_result, sizeof(double)));
	checkCudaErrors(hipMalloc(&d_gradEqls, sizeof(EqInfo) * gradEqs.size()));
	checkCudaErrors(hipMalloc(&d_objEqls, sizeof(EqInfo) * objEqs.size()));
	checkCudaErrors(hipMalloc(&d_gradEqHeads, sizeof(int) * gradEqHeads.size()));
	checkCudaErrors(hipMalloc(&d_objEqHeads, sizeof(int) * objEqHeads.size()));
	checkCudaErrors(hipMalloc(&d_gradEqVals, sizeof(double) * gradEqVals.size()));
	checkCudaErrors(hipMalloc(&d_objEqVals, sizeof(double) * objEqVals.size()));
	checkCudaErrors(hipMalloc(&d_fprev, sizeof(double)));
	checkCudaErrors(hipMalloc(&d_fnow, sizeof(double)));
	checkCudaErrors(hipMalloc(&tmp, sizeof(double) * numObjEqs));
	checkCudaErrors(hipMemcpy(d_gradEqHeads, gradEqHeads.data(), sizeof(int) * gradEqHeads.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_objEqHeads, objEqHeads.data(), sizeof(int) * objEqHeads.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_gradEqls, gradEqs.data(), sizeof(EqInfo) * gradEqs.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_objEqls, objEqs.data(), sizeof(EqInfo) * objEqs.size(), hipMemcpyHostToDevice));

	int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK/32) + 1);
	int numBlocksPerSm = 0;
	int numThreads = THREADS_PER_BLOCK;

	checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
		&numBlocksPerSm, BFGSMultiply, numThreads, sMemSize));

	int numSms = deviceProp.multiProcessorCount;
	dim3 dimGrid(numSms * numBlocksPerSm, 1, 1),
		dimBlock(THREADS_PER_BLOCK, 1, 1);

	int allnum = gradEqVals.size();
	int allnumObj = objEqVals.size();

	fPrev = _CalcObj(xNow, objEqs, numObjEqs);
	_CalcGrad(xNow, gPrev, gradEqs);
	checkCudaErrors(hipMemcpy(d_gprev, gPrev.data(), sizeof(double) * n, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_xnow, xNow.data(),	sizeof(double) * n, hipMemcpyHostToDevice));
	
	bool exit_con = false;
	while(!exit_con)
	{
		void* kernelArgsInit[] = {
			(void*)&d_H,
			(void*)&d_p,
			(void*)&d_gprev,
			(void*)&d_dot_result,
			(void*)&n
		};
		
		dim3 dimGrid1(16,1,1), dimBlock1(32,1,1);
		checkCudaErrors(hipLaunchCooperativeKernel((void *)gpuInitHp,
													dimGrid1, dimBlock1, kernelArgsInit,
													sMemSize, NULL));

		while(true)
		{
			if (itCounter++ > itMax)
			{
				exit_con = true;
				break;
			}
			
			checkCudaErrors(hipMemcpy(d_xprev, d_xnow, sizeof(double) * n, hipMemcpyDeviceToDevice));


			// if (fNow > fPrev) {
			// 	_VecCopy(xNow, xPrev);
			// 	break;
			// }

			// if (k == n) {
			// 	fPrev = fNow;
			// 	_VecCopy(gPrev, gNow);
			// 	break;
			// }
			checkCudaErrors(hipMemset(d_sy, 0, sizeof(double)));

			void *kernelArgsIntervel[] = {
					(void*)&d_xprev, (void*)&step, (void*)&d_p, (void*)&d_xt, (void*)&n, (void*)&left, (void*)&right, 
					(void*)&d_dot_result, (void*)&tmp, (void*)&d_objEqls, (void*)&numObjEqs, (void*)&allnumObj, (void*)&d_objEqHeads, (void*)&d_objEqHeads 
			};

			void *kernelArgsSep[] = {
					(void*)&d_xprev, (void*)&d_p, (void*)&d_xt, (void*)&d_xnow, (void*)&n, (void*)&left, (void*)&right, 
					(void*)&d_dot_result, (void*)&tmp, (void*)&d_objEqls, (void*)&numObjEqs, (void*)&allnumObj, (void*)&d_objEqHeads, (void*)&d_objEqVals 
			};
			
			dim3 dimGrid1(16,1,1), dimBlock1(32,1,1);
			checkCudaErrors(hipLaunchCooperativeKernel((void *)gpuDetermineInterval,
														dimGrid1, dimBlock1, kernelArgsIntervel,
														sMemSize, NULL));

			checkCudaErrors(hipLaunchCooperativeKernel((void *)gpuGodenSep,
														dimGrid1, dimBlock1, kernelArgsSep,
														sMemSize, NULL));
														
			void *kernelArgsObj[] = {
					(void*)&d_xnow, (void*)&tmp, (void*)&numObjEqs, (void*)&allnumObj, (void*)&d_objEqHeads, 
					(void*)&d_objEqls, (void*)&d_objEqVals, (void*)&d_fnow 
			};

			checkCudaErrors(hipLaunchCooperativeKernel((void *)gpuCalcObj,
														dimGrid, dimBlock, kernelArgsObj,
														sMemSize, NULL));
			
			checkCudaErrors(hipMemcpy(&fNow, d_fnow, sizeof(double), hipMemcpyDeviceToHost));
			
			std::cout << itCounter << " iterations, " <<	"f(x) = " << fNow << std::endl;

			if (fNow < eps)
			{
				exit_con = true;
				break;
			}

			gpuCalcGrad <<< 64, 64 >>>(d_xnow, d_gnow, n,  gradEqVals.size(), d_gradEqHeads, d_gradEqls, d_gradEqVals);

			void *kernelArgs[] = {
					(void*)&d_gprev, (void*)&d_gnow, (void*)&d_xprev, (void*)&d_xnow, 
					(void*)&d_H, (void*)&d_p, (void*)&d_yTH, (void*)&d_Hy, (void*)&d_s,
					(void*)&d_y, (void*) &fPrev, (void*)&fNow, (void*)&d_sy, (void*)&d_dot_result, (void*)&n,
					(void*)&allnum, (void*)&d_gradEqHeads, (void*)&d_gradEqls, (void*)&d_gradEqVals,
			};

			checkCudaErrors(hipLaunchCooperativeKernel((void *)BFGSMultiply,
														dimGrid, dimBlock, kernelArgs,
														sMemSize, NULL));
			fPrev = fNow;

			checkCudaErrors(hipMemcpy(d_gprev, d_gnow, sizeof(double) * n, hipMemcpyDeviceToDevice));
			checkCudaErrors(hipMemcpy(d_xprev, d_xnow, sizeof(double) * n, hipMemcpyDeviceToDevice));
			double sy = 0;
			checkCudaErrors(hipMemcpy(&sy, d_sy, sizeof(double), hipMemcpyDeviceToHost));

			if(sy < epsZero1)
			{
				exit_con = true;
				break;
			}
		}
	}

	std::cout << itCounter << " iterations" << std::endl;
	std::cout << "f(x) = " << fNow << std::endl;
	double dt = omp_get_wtime()-t0;
	printf("###Solver totally used %2.5f s ...\n", dt);

	//Put results back...
	if (fNow < eps) {
		printf("Solved!!!!\n");
		return true;
	}
	else {
		printf("Solver Failed!!!!\n");
		return false;
	}

	checkCudaErrors(hipFree(d_gnow));
	checkCudaErrors(hipFree(d_gprev));
	checkCudaErrors(hipFree(d_xnow));
	checkCudaErrors(hipFree(d_xprev));
	checkCudaErrors(hipFree(d_s));
	checkCudaErrors(hipFree(d_y));
	checkCudaErrors(hipFree(d_yTH));
	checkCudaErrors(hipFree(d_Hy));
	checkCudaErrors(hipFree(d_p));
	checkCudaErrors(hipFree(d_H));
	checkCudaErrors(hipFree(d_sy));
	checkCudaErrors(hipFree(d_dot_result));
	checkCudaErrors(hipFree(d_gradEqls));
	checkCudaErrors(hipFree(d_objEqls));
	checkCudaErrors(hipFree(d_gradEqHeads));
	checkCudaErrors(hipFree(d_objEqHeads));
	checkCudaErrors(hipFree(d_gradEqVals));
	checkCudaErrors(hipFree(d_objEqVals));
	checkCudaErrors(hipFree(d_fnow));
	checkCudaErrors(hipFree(tmp));
}

int main()
{
	BFGSSolveEqs();
}

